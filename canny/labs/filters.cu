#include "hip/hip_runtime.h"
// To calculate the Grayscale image = ColorToGrayscale
// For the gaussian blur = Conv2D
// and the sobel filter which gives the 
//  gradient descent = GradientSobel
#include "filters.h"

void populate_blur_filter(double *outFilter, size_t filterEdgeLen)
{
    //double scaleVal = 1;
    //double stDev = (double)FILTERSIZE/3;

    double stDevSq = 0.6;
    double pi = M_PI;
	 double scaleFac = (1 / (2*pi*stDevSq));

    for (int i = 0; i < filterEdgeLen; ++i) {
        for (int j = 0; j < filterEdgeLen; ++j) {

			// pow() is slow so just multiply out
            double xComp = (i + 1 - (filterEdgeLen+1)/2) * (i + 1 - (filterEdgeLen+1)/2);
            double yComp = (j + 1 - (filterEdgeLen+1)/2) * (j + 1 - (filterEdgeLen+1)/2);

            //calculate the value at each index of the Kernel
            double filterVal = exp(-(xComp + yComp) / (2 * stDevSq));
            filterVal = scaleFac * filterVal;

            //populate Kernel
            outFilter[i + j*filterEdgeLen] = filterVal;

	/*
            if (i==0 && j==0)
            {
                scaleVal = outFilter[0][0];
            }

            //normalize Kernel
            outFilter[i][j] = outFilter[i][j] / scaleVal;
	*/			

        }
    }
}


void ColorToGrayscaleSerial(float *input, float *output,
                    unsigned int y, unsigned int x) {
  for (unsigned int ii = 0; ii < y; ii++) {
    for (unsigned int jj = 0; jj < x; jj++) {
      unsigned int idx = ii * x + jj;
      float r          = input[3 * idx];     // red value for pixel
      float g          = input[3 * idx + 1]; // green value for pixel
      float b          = input[3 * idx + 2];
      output[idx] = (float)(0.21f * r + 0.71f * g + 0.07f * b);
    }
  }
}

// convert the image to grayscale
__global__ void ColorToGrayscale(float *inImg, float *outImg, int width, int height) {
   int idx, grayidx;
   int col = blockDim.x * blockIdx.x + threadIdx.x;
   int row  = blockDim.y * blockIdx.y + threadIdx.y;
   int numchannel = 3;

   // x = col and y = row
   if (col >= 0 && col < width && row >=0 && row < height) {
      // each spot is 3 big (rgb) so get the number of spots
      grayidx = row * width + col;
      idx     = grayidx * numchannel; // and multiply by three
      // to calculate the beginning of the 3 for that pixel
      float r = inImg[idx];           //red
      float g = inImg[idx + 1];       //green
      float b = inImg[idx + 2];       //blue
      outImg[grayidx]  = (0.21*r + 0.71*g + 0.07*b);
   }
}



// the gaussian blur is just a conv2d with a filter
__global__ void Conv2D(float *inImg, float *outImg, double *filter, int width, int height, size_t filterSize) {
   int row = blockIdx.y * blockDim.y + threadIdx.y;
   int col = blockIdx.x * blockDim.x + threadIdx.x;
   int halfFilter = (int)filterSize/2;

   // boundary check if it's in the image
   if(row > 0 && row < height && col > 0 && col < width) {
      float pixelvalue = 0;
      int start_col = col - halfFilter;
      int start_row = row - halfFilter;
      
      // now do the filtering
      for (int j = 0; j < filterSize; ++j) {
         for (int k = 0; k < filterSize; ++k) {
	    int cur_row = start_row + j;
            int cur_col = start_col + k;
           
            // only count the ones that are inside the boundaries
            if (cur_row >=0 && cur_row < height && cur_col >= 0 && cur_col < width) {
               pixelvalue += inImg[cur_row*width + cur_col] * filter[j + k*filterSize];
	    }
           
         }
      }
      __syncthreads();
      outImg[row*width + col] = pixelvalue;      
   }

}

__global__ void GradientSobel(float *inImg, float *sobelImg, float *gradientImg, int height, int width, size_t filterSize) {
  //int filterSize = (int)FILTERSIZE;
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  // To detect horizontal lines, G_x. 
  const int fmat_x[3][3] = {
    {-1, 0, 1},
    {-2, 0, 2},
    {-1, 0, 1}
  };
  // To detect vertical lines, G_y 
  const int fmat_y[3][3]  = {
    {-1, -2, -1},
    {0,   0,  0},
    {1,   2,  1}
  };

  // now do the filtering
  // halfFitler is how many are on each side
  int halfFilter = (int)filterSize/2;
  double sumx = 0;
  double sumy = 0;
  //// DO THE SOBEL FILTERING ///////////

  // boundary check if it's in the image
  if(row > 0 && row < height && col > 0 && col < width) {
    int start_col = col - halfFilter;
    int start_row = row - halfFilter;

    // now do the filtering
    for (int j = 0; j < filterSize; ++j) {
      for (int k = 0; k < filterSize; ++k) {
        int cur_row = start_row + j;
        int cur_col = start_col + k;

        // only count the ones that are inside the boundaries
        if (cur_row >=0 && cur_row < height) {
          sumy += inImg[cur_row*width + cur_col] * fmat_y[j][k];
        }
        if ( cur_col >= 0 && cur_col < width) {
          sumx += inImg[cur_row*width + cur_col] * fmat_x[j][k];
        }
        __syncthreads();
      }
    }
    // now calculate the sobel output and gradients
    int value = sqrt(sumx * sumx + sumy*sumy);
    if (value > 255) {
      value = 255;
    } 
    if (value < 0) {
      value = 0;
    }
    sobelImg[row*width + col] = value; // output of the sobel filter
    gradientImg[row*width + col] = atan(sumx/sumy) * 180/M_PI; // the gradient calculateion
  }

  //__syncthreads();

}


void Conv2DSerial(float *inImg, float *outImg, double *filter, int width, int height, size_t filterSize) {

    // find center position of kernel (half of kernel size)
    int filterHalf = filterSize / 2;
    
    // iterate over rows and coluns of the image
    for(int row=0; row < height; ++row)              // rows
    {
        for(int col=0; col < width; ++col)          // columns
        {
            int start_col = col - filterHalf;
            int start_row = row - filterHalf;
            float pixelvalue = 0; 

            // then for each pixel iterate through the filter
            for(int j=0; j < filterSize; ++j)     // filter rows
            {
                for(int k=0; k < filterSize; ++k) // kernel columns
                {
                    int cur_row = start_row + j;
                    int cur_col = start_col + k;
                    if (cur_row >= 0 && cur_row < height && cur_col >= 0 && cur_col < width) {
                        pixelvalue += inImg[cur_row*width + cur_col] * filter[j + k*filterSize];
                    }
                }
            }
            outImg[row*width+col] = pixelvalue;
        }
    }
}


void GradientSobelSerial(float *inImg, float *mag, float *phase, int height, int width, size_t filterSize)
{

	//int filterSize = (int)FILTERSIZE;
	int halfFilter = (int)(filterSize)/2;
	
	// To detect horizontal lines, G_x. 
	const int fmat_x[3][3] = {
		{-1, 0, 1},
		{-2, 0, 2},
		{-1, 0, 1}
	};
	// To detect vertical lines, G_y 
	const int fmat_y[3][3]  = {
		{-1, -2, -1},
		{0,   0,  0},
		{1,   2,  1}
	};

	// iterate over rows and columns of the image
	for(int row=0; row < height; ++row)              // rows
	{
		for(int col=0; col < width; ++col)          // columns
		{

			double sumx = 0;
			double sumy = 0;

			int start_col = col - halfFilter;
			int start_row = row - halfFilter;

			// now do the filtering
			for (int j = 0; j < filterSize; ++j)
			{
				for (int k = 0; k < filterSize; ++k)
				{

					int cur_row = start_row + j;
					int cur_col = start_col + k;

					// only count the ones that are inside the boundaries
					if (cur_row >= 0 && cur_row < height && cur_col >= 0 && cur_col < width)
					{
						sumy += inImg[cur_row*width + cur_col] * fmat_y[j][k];
						sumx += inImg[cur_row*width + cur_col] * fmat_x[j][k];
					}

				}
			}

			mag[row*width + col] = sqrt(sumx * sumx + sumy * sumy); // output of the sobel filt at this index
			phase[row*width+col] = atan(sumx/sumy) * 180/M_PI; // gradient at pixel

		}
	}
}
